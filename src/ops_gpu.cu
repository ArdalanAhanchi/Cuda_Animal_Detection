#include "hip/hip_runtime.h"
/**
 *  An implementation for a class which implements the ops interface for basic matrix 
 *  operations on the gpu. This is used as a total gpu implementation.
 *
 *  @author Ardalan Ahanchi
 *  @date March 2020
 */

#include "ops_gpu.cuh"
#include "mat.hpp"

#include ""
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>                                                        //For exponents.

//Define a device number if not defined in compilation.
#ifndef DEVICE_NUM
#define DEVICE_NUM 0
#endif

//Uses a TILE x TILE block for tiling. It can support up to 32x32 tiles.
#ifndef TILE 
#define TILE 8
#endif

//Number of threads launched per block for simple functions (all except matrix mult).
#define TPB 256

namespace anr {

/** The offset index used for each naive kernel (since the kernel exectues many times). */
__constant__ size_t kernel_offset[1];

/** The constant value which is used for the scale kernel. */
__constant__ type scale_number[1];

//Thes values are only used for matrix multiplication.
__constant__ size_t a_dims[2];  /**< Number of rows [0], and cols [1] in first matrix. */
__constant__ size_t b_dims[2];  /**< Number of rows [0], and cols [1] in second matrix. */
__constant__ size_t c_dims[2];  /**< Number of rows [0], and cols [1] in output matrix. */


/**
 *  A kernel which performs addition on the gpu. it adds every element of a and b and
 *  stores it in matrix c. It also takes into account the kerenel offset.
 *
 *  @param a_gpu location of the first matrix on GPU.
 *  @param b_gpu location of the second matrix on GPU.
 *  @param c_gpu location of the output matrix on GPU.
 */
__global__ void add_kernel(type* a_gpu, type* b_gpu, type* c_gpu) {
    //Calculate the current index in all the matrices and perform the addition.
    size_t idx = kernel_offset[0] + threadIdx.x + (blockIdx.x * blockDim.x);
    c_gpu[idx] = a_gpu[idx] + b_gpu[idx];
}


/**
 *  A kernel which performs subtraction on the gpu. it subtracts every element of 
 *  b from a and stores it in matrix c. It also takes into account the kerenel offset.
 *
 *  @param a_gpu location of the first matrix on GPU.
 *  @param b_gpu location of the second matrix on GPU.
 *  @param c_gpu location of the output matrix on GPU.
 */
__global__ void sub_kernel(type* a_gpu, type* b_gpu, type* c_gpu) {
    //Calculate the current index in all the matrices and perform the subtraction.
    size_t idx = kernel_offset[0] + threadIdx.x + (blockIdx.x * blockDim.x);
    c_gpu[idx] = a_gpu[idx] - b_gpu[idx];
}


/**
 *  A kernel which performs element by element multiplication on the gpu. it multiplies 
 *  every element of a into b and stores it in matrix c. It also takes into account the 
 *  kerenel offset.
 *
 *  @param a_gpu location of the first matrix on GPU.
 *  @param b_gpu location of the second matrix on GPU.
 *  @param c_gpu location of the output matrix on GPU.
 */
__global__ void e_mult_kernel(type* a_gpu, type* b_gpu, type* c_gpu) {
    //Calculate the current index in all the matrices and perform the multiplication.
    size_t idx = kernel_offset[0] + threadIdx.x + (blockIdx.x * blockDim.x);
    c_gpu[idx] = a_gpu[idx] * b_gpu[idx];
}


/**
 *  A kernel which performs tiled matrix multiplication on the gpu. It also uses shared
 *  memory and it can perform on arbitrary sized matrices.
 *
 *  @param a_gpu location of the first matrix on GPU.
 *  @param b_gpu location of the second matrix on GPU.
 *  @param c_gpu location of the output matrix on GPU.
 */
__global__ void mult_kernel(type* a_gpu, type* b_gpu, type* c_gpu) {
    //Holds each tile with TILE_SIDE x TILE_SIDE size for A and B matrices.
    __shared__ type a_tile[TILE * TILE];
	__shared__ type b_tile[TILE * TILE];

    //Seperately stored for caching, and simplifying the complex indexing.
	int tx = threadIdx.x;
    int ty = threadIdx.y;

	//Find the row and column for the output matrix (C).
	int row = blockIdx.x * TILE + tx;
	int col = blockIdx.y * TILE + ty;

    //Used to find C at the end of calculations within the block.
	type sum_c = 0;

	//Go through every tile (in a single direction).
	for(int t = 0; t < ((a_dims[0] - 1) / TILE + 1) ; t++)
    {
        //Load from A matrix into shared memory tile (if not a boundry).
        a_tile[(tx * TILE) + ty] = 
            (row < a_dims[0] && ((t * TILE) + ty) < a_dims[1]) ? 
            a_gpu[(row * a_dims[1]) + t * TILE + ty] : 0.0;

        //Load from B matrix into shared memory tile (if not a boundry).
        b_tile[(tx * TILE) + ty] = 
            (col < b_dims[1] && ((t * TILE) + tx) < b_dims[0]) ?
            b_gpu[col + ((t * TILE) + ty) * b_dims[1]] : 0.0;

		//Wait till all the data is completely stored in shared memory.
		__syncthreads();

        //Multiply the values from the matrices stored in shared memory.
		for(int i = 0; i < TILE; i++)
			sum_c += a_tile[(tx * TILE) + i] * b_tile[(i * TILE) + ty];

		//Wait till the sum is calculated before adding it up for the assignment.
		__syncthreads();
	}

	//Check if the current row/col are within the range, and then assign the value.
	if(row < c_dims[0] && col < c_dims[1])
		c_gpu[row * c_dims[1] + col] = sum_c;
}


/**
 *  A kernel which performs scaling of every matrix element and stores it in outputs.
 *  the amount of scaling is stored in a GPU constant called scale.
 *
 *  @param input location of the input matrix on GPU.
 *  @param input location of the input matrix on GPU.
 */
__global__ void scale_kernel(type* input, type* output) {
    //Calculate the current index in the matrix and perform the scaling.
    size_t idx = kernel_offset[0] + threadIdx.x + (blockIdx.x * blockDim.x);
    output[idx] = input[idx] * scale_number[0];
}


/**
 *  A kernel which performs a sigmoid for every element of the matrix in-place 
 *  and overwrites the current element of the matrix.
 *
 *  @param input location of the input matrix on GPU.
 */
__global__ void sigmoid_kernel(type* input) {
    //Calculate the current index in the matrix and perform the function.
    size_t idx = kernel_offset[0] + threadIdx.x + (blockIdx.x * blockDim.x);
    input[idx] = 1.0 / (1.0 + exp(-input[idx]));
}


/**
 *  A kernel which performs a sigmoid prime for every element of the matrix in-place 
 *  and overwrites the current element of the matrix.
 *
 *  @param input location of the input matrix on GPU.
 */
__global__ void d_sigmoid_kernel(type* input) {
    //Calculate the current index in the matrix and perform the function.
    size_t idx = kernel_offset[0] + threadIdx.x + (blockIdx.x * blockDim.x);
    input[idx] = exp(-input[idx]) / pow((1.0 + exp(-input[idx])), 2);
}


/**
 *  A kernel which performs a relu for every element of the matrix in-place 
 *  and overwrites the current element of the matrix.
 *
 *  @param input location of the input matrix.
 */
__global__ void relu_kernel(type* input) {
    //Calculate the current index in the matrix and perform the function.
    size_t idx = kernel_offset[0] + threadIdx.x + (blockIdx.x * blockDim.x);

    //Perform the relu function.
    if(input[idx] < 0.0)
        input[idx] = 0.0;
}


/**
 *  A kernel which performs a relu prime for every element of the matrix in-place 
 *  and overwrites the current element of the matrix.
 *
 *  @param input location of the input matrix on GPU.
 */
__global__ void d_relu_kernel(type* input) {
    //Calculate the current index in the matrix.
    size_t idx = kernel_offset[0] + threadIdx.x + (blockIdx.x * blockDim.x);

    //Perform the relu prime function.
    if (input[idx] <= 0.0)
        input[idx] = 0.0;
    else
        input[idx] = 1.0;
}


/**
 *  A function which adds matrix a and b, and returns a results matrix. a, and b 
 *  should be of exactly the same size. Should return a 0x0 matrix if error occured.
 *
 *  @param a The first matrix for the addition.
 *  @param b The second matrix for the addition.
 *  @return The results for the addition (only the pointer is passed by value).
 */
Mat Ops_gpu::add(const Mat& a, const Mat& b) {
    //Check if the matrices not the same size, print an error and return a 0x0 mat.
    if(a.rows() != b.rows() || a.cols() != b.cols())
        return error("Matrices should be the same dimentions", Op_Code::_add);

    //Define the output matrix to be the same size as a and b.
    Mat output(a.rows(), a.cols());

    //Transfer the input matrices to the GPU.
    type* a_gpu = this->transfer_to_gpu(a);
    type* b_gpu = this->transfer_to_gpu(b);

    //Allocate some memory on the GPU for the output matrix based on the size.
    type* output_gpu;
    size_t output_size = output.rows() * output.cols();
    hipMalloc((void**) &output_gpu, ((size_t) sizeof(type)) * output_size);

    //Call the operations function with the correct opcode.
    this->operation(Op_Code::_add, output_size, a_gpu, b_gpu, output_gpu);

    //Transfer the data back into the output matrix.
    this->transfer_from_gpu(output_gpu, output);

    //Deallocate the memory on the GPU.
    hipFree(a_gpu);
    hipFree(b_gpu);
    hipFree(output_gpu);

    //Return the results.
    return output;
}


/**
 *  A function which subtracts matrix a and b, and returns a results matrix. a, and b 
 *  should be of exactly the same size. It basically returns a - b. Should return a 
 *  0x0 matrix if error occured.
 *
 *  @param a The first matrix for the subtraction.
 *  @param b The second matrix for the subtraction.
 *  @return The results for the subtraction (only the pointer is passed by value).
 */
Mat Ops_gpu::sub(const Mat& a, const Mat& b) {
    //Check if the matrices not the same size, print an error and return a 0x0 mat.
    if(a.rows() != b.rows() || a.cols() != b.cols())
        return error("Matrices should be the same dimentions", Op_Code::_sub);

    //Define the output matrix to be the same size as a and b.
    Mat output(a.rows(), a.cols());

    //Transfer the input matrices to the GPU.
    type* a_gpu = this->transfer_to_gpu(a);
    type* b_gpu = this->transfer_to_gpu(b);

    //Allocate some memory on the GPU for the output matrix based on the size.
    type* output_gpu;
    size_t output_size = output.rows() * output.cols();
    hipMalloc((void**) &output_gpu, ((size_t) sizeof(type)) * output_size);

    //Call the operations function with the correct opcode.
    this->operation(Op_Code::_sub, output_size, a_gpu, b_gpu, output_gpu);

    //Transfer the data back into the output matrix.
    this->transfer_from_gpu(output_gpu, output);

    //Deallocate the memory on the GPU.
    hipFree(a_gpu);
    hipFree(b_gpu);
    hipFree(output_gpu);

    //Return the results.
    return output;
}


/**
 *  A function which performs an element by element multiplication of matrices a 
 *  and b. The matrices should be the same size.
 *
 *  @param a The first matrix for the matrix multiplication.
 *  @param b The second matrix for the matrix multiplication.
 *  @return The results for the multiplication (only the pointer is passed by value).
 */
Mat Ops_gpu::e_mult(const Mat& a, const Mat& b) {
    //Check if the matrices not the same size, print an error and return a 0x0 mat.
    if(a.rows() != b.rows() || a.cols() != b.cols())
        return error("Matrices should be the same dimentions", Op_Code::_e_mult);

    //Define the output matrix to be the same size as a and b.
    Mat output(a.rows(), a.cols());

    //Transfer the input matrices to the GPU.
    type* a_gpu = this->transfer_to_gpu(a);
    type* b_gpu = this->transfer_to_gpu(b);

    //Allocate some memory on the GPU for the output matrix based on the size.
    type* output_gpu;
    size_t output_size = output.rows() * output.cols();
    hipMalloc((void**) &output_gpu, ((size_t) sizeof(type)) * output_size);

    //Transfer the data back into the output matrix.
    this->transfer_from_gpu(output_gpu, output);

    //Deallocate the memory on the GPU.
    hipFree(a_gpu);
    hipFree(b_gpu);
    hipFree(output_gpu);

    //Return the results.
    return output;
}


/**
 *  A function which multiplies matrix a and b, and returns a results matrix. a should
 *  have the same number of cols, as b's rows. It basically returns a * b. Should
 *  return a 0x0 matrix if error occured.
 *
 *  @param a The first matrix for the matrix multiplication.
 *  @param b The second matrix for the matrix multiplication.
 *  @return The results for the multiplication (only the pointer is passed by value).
 */
Mat Ops_gpu::mult(const Mat& a, const Mat& b) {
    //Check if we have invalid sizes for the matrices.
    if(a.cols() != b.rows())
        return error("Invalid sizes for matrix multiplication", Op_Code::_mult);

    //Define the output matrix to be the correct size for output.
    Mat c(a.rows(), b.cols());

    //Transfer the input matrices to the GPU.
    type* a_gpu = this->transfer_to_gpu(a);
    type* b_gpu = this->transfer_to_gpu(b);

    //Allocate some memory on the GPU for the output matrix based on the size.
    type* c_gpu;
    size_t c_size = c.rows() * c.cols();
    hipMalloc((void**) &c_gpu, ((size_t) sizeof(type)) * c_size);

    //Store the dimetions in arrays for transfering to constant memory.
    size_t a_dims_host[2] = {a.rows(), a.cols()};
    size_t b_dims_host[2] = {b.rows(), b.cols()};
    size_t c_dims_host[2] = {c.rows(), c.cols()};

    //Store the dimentions of the matrices in constant memory in gpu.
    hipMemcpyToSymbol(HIP_SYMBOL(a_dims), a_dims_host, sizeof(size_t) * 2);
    hipMemcpyToSymbol(HIP_SYMBOL(b_dims), b_dims_host, sizeof(size_t) * 2);
    hipMemcpyToSymbol(HIP_SYMBOL(c_dims), c_dims_host, sizeof(size_t) * 2);

    //Determine the size for 2D grids, and blocks using the tile sizes.    
    dim3 threads(TILE, TILE);

    //Side of the grid is rows/TILE_SIDE x cols/TILE_SIDE.
    dim3 blocks(std::ceil((type) c.rows() / (type) TILE), 
        std::ceil((type) c.cols() / (type) TILE));

    //Run the kernel with the calculated number of blocks and threads.
    mult_kernel<<<blocks, threads>>>(a_gpu, b_gpu, c_gpu);

    //Transfer the data back into the output matrix.
    this->transfer_from_gpu(c_gpu, c);

    //Deallocate the memory on the GPU.
    hipFree(a_gpu);
    hipFree(b_gpu);
    hipFree(c_gpu);

    //Return the results.
    return c;
}


/**
 *  A function which scales the matrix a by a scalar value. It Should
 *  return a 0x0 matrix if error occured.
 *
 *  @param a The matrix which we're scaling.
 *  @param scale The scalar value which is multiplied to every element of a.
 *  @return The results for the scaling (only the pointer is passed by value).
 */ 
Mat Ops_gpu::scale(const Mat& a, const type& scale) {
    //Define the output matrix to be the same size as a.
    Mat output(a.rows(), a.cols());

    //Transfer the input matrix to the GPU.
    type* a_gpu = this->transfer_to_gpu(a);

    //Allocate some memory on the GPU for the output matrix based on the size.
    type* output_gpu;
    size_t output_size = output.rows() * output.cols();
    hipMalloc((void**) &output_gpu, ((size_t) sizeof(type)) * output_size);

    //Store the scale value on the GPU's constant memory.
    type scale_host[1] = { (scale) };
    hipMemcpyToSymbol(HIP_SYMBOL(scale_number), scale_host, sizeof(type));

    //Call the operations function with the correct opcode.
    this->operation(Op_Code::_scale, output_size, a_gpu, nullptr, output_gpu);

    //Transfer the data back into the output matrix.
    this->transfer_from_gpu(output_gpu, output);

    //Deallocate the memory on the GPU.
    hipFree(a_gpu);
    hipFree(output_gpu);

    //Return the output matrix.
    return output;
}


/**
 *  A method which applies the sigmoid function to the passed matrix.
 *
 *  @param input The matrix where we're applying the sigmoid to. 
 */
void Ops_gpu::sigmoid(Mat& input) {
    //Transfer the input matrix to the GPU.
    type* input_gpu = this->transfer_to_gpu(input);
    size_t input_size = input.rows() * input.cols();

    //Call the operations function with the correct opcode.
    this->operation(Op_Code::_sigmoid, input_size, input_gpu);

    //Transfer the data back into the input matrix and override it.
    this->transfer_from_gpu(input_gpu, input);

    //Deallocate the GPU allocation.
    hipFree(input_gpu);
} 


/**
 *  A method which applies the derivative of sigmoid function to the passed matrix.
 *
 *  @param input The matrix where we're applying the sigmoid to. 
 */
void Ops_gpu::deriv_sigmoid(Mat& input) {
    //Transfer the input matrix to the GPU.
    type* input_gpu = this->transfer_to_gpu(input);
    size_t input_size = input.rows() * input.cols();

    //Call the operations function with the correct opcode.
    this->operation(Op_Code::_d_sigmoid, input_size, input_gpu);

    //Transfer the data back into the input matrix and override it.
    this->transfer_from_gpu(input_gpu, input);

    //Deallocate the GPU allocation.
    hipFree(input_gpu);
}


/**
 *  A method which applies the relu function to the passed matrix.
 *
 *  @param input The matrix where we're applying the relu to.
 */
void Ops_gpu::relu(Mat& input) {
    //Transfer the input matrix to the GPU.
    type* input_gpu = this->transfer_to_gpu(input);
    size_t input_size = input.rows() * input.cols();

    //Call the operations function with the correct opcode.
    this->operation(Op_Code::_relu, input_size, input_gpu);

    //Transfer the data back into the input matrix and override it.
    this->transfer_from_gpu(input_gpu, input);

    //Deallocate the GPU allocation.
    hipFree(input_gpu);
}


/**
 *  A method which applies the derivative of the relu function.
 *
 *  @param input The matrix where we're applying the relu to.
 */
void Ops_gpu::deriv_relu(Mat& input) {
    //Transfer the input matrix to the GPU.
    type* input_gpu = this->transfer_to_gpu(input);
    size_t input_size = input.rows() * input.cols();

    //Call the operations function with the correct opcode.
    this->operation(Op_Code::_d_relu, input_size, input_gpu);

    //Transfer the data back into the input matrix and override it.
    this->transfer_from_gpu(input_gpu, input);

    //Deallocate the GPU allocation.
    hipFree(input_gpu);
}


/**
 *  A function which transfers the given matrix to the GPU for calculations.
 *
 *  @param input The matrix object which we're transferring.
 *  @return The returned GPU address for the allocated memory.
 */
type* Ops_gpu::transfer_to_gpu(const Mat& input) {
    //Calculate the memory size based on the size of matrix.
    size_t memory_size = ((size_t)sizeof(type)) * input.rows() * input.cols();
    type* output; 
    hipMalloc((void**) &output, memory_size);

    //Copy the data from the CPU to GPU.
    hipMemcpy(output, input.data, memory_size, hipMemcpyHostToDevice);

    //Return a pointer to the memory on GPU.
    return output;
}


/**
 *  A function which transfers back the results from the GPU after calculations.
 *
 *  @param source The GPU memory location for the transfer.
 *  @param output The destination host mat object used to write the data.
 */
void Ops_gpu::transfer_from_gpu(type* source, Mat& output) {
    //Calculate the memory size based on the size of output matrix.
    size_t memory_size = ((size_t)sizeof(type)) * output.rows() * output.cols();

    //Trasfer the matrix back to the host.
    hipMemcpy(output.data, source, memory_size, hipMemcpyDeviceToHost);
}


/**
 *  A function which performs simple cuda operations based on a given opcode.
 *  It accepts up to 3 matrices, and it will call the appropriate function based on
 *  the kernel_call function. Matrices b, and c could be nullptr for some opcodes.
 *
 *  @param opcode The operation code based on the opcode struct.
 *  @param output_size The number of elements in the output matrix (either a or c).
 *  @param a A pointer to the location of the first input matrix on GPU.
 *  @param b A pointer to the location of the second input matrix on GPU.
 *  @param c A pointer to the location of the output matrix on GPU.
 */
void Ops_gpu::operation(size_t opcode, size_t output_size, 
    type* a_gpu, type* b_gpu, type* c_gpu) {

    //Get the device properties for optimizing the memory usage.
    hipDeviceProp_t stats;
    hipGetDeviceProperties(&stats, DEVICE_NUM);

    //Get the maximum number of threads, and blocks based on the current architecture.
    size_t max_threads = (size_t) stats.maxThreadsPerBlock;
    size_t max_blocks = (size_t) stats.maxGridSize[0];

    //Select the correct number of threads based on the maximum supported, and requested.
    max_threads = (TPB > max_threads ? max_threads : TPB);

    //Holds the total number of calculations we need.
    size_t num_calcs = output_size;

    //Go through and calculate the kernel untill we finish the calculation.
    while(num_calcs != 0) {
        //Find the number of threads needed (based on how many calculations we have left).
        size_t num_threads = (num_calcs > max_threads ? max_threads : num_calcs);

        //Calculate the number of blocks required.
        size_t num_blocks = std::floor(num_calcs / num_threads);

        //Check if the number of blocks is more than maximum supported on the architecture.
        if(num_blocks > max_blocks)
            num_blocks = max_blocks;

        //Save the kernel offset in constant memory.
        size_t kernel_offset_host[1] = { (output_size - num_calcs) };
        hipMemcpyToSymbol(HIP_SYMBOL(kernel_offset), kernel_offset_host, sizeof(size_t));

        //Perform the kernel call based on the opcode.
        this->kernel_call(opcode, num_blocks, num_threads, a_gpu, b_gpu, c_gpu);

        //Reduce the number of calculations left based on the amount calculated.
        num_calcs -= (num_blocks * num_threads);
    }
}


/**
 *  A function which prints an error message (passed to it), and then returns a 0x0
 *  matrix (to be returned by the calling function. It also accepts an opcode which
 *  it uses for printing out errors in a nice format.
 *
 *  @param msg The message which will be printed.
 *  @param opcode The opcode for the calling function.
 *  @return A 0x0 matrix which can be returned by the function.
 */
Mat Ops_gpu::error(std::string msg, size_t opcode) {
    std::cerr << "Ops_gpu: " << Op_Names[opcode] << ": Error: " << msg << std::endl;
    return Mat(0, 0);
}


/**
 *  A function which calls the appropriate kernel for the opcode.
 *
 *  @param opcode The opcode corresponding to the kernel.
 *  @param blocks The number of blocks used for the execution.
 *  @param threads The number of threads used for the execution.
 *  @param a A pointer to the location of the first input matrix on GPU.
 *  @param b A pointer to the location of the second input matrix on GPU.
 *  @param c A pointer to the location of the output matrix on GPU.
 */
void Ops_gpu::kernel_call(size_t opcode, size_t blocks, size_t threads, 
    type* a, type* b, type* c) {

    //Perform the operation based on the opcode.
    switch(opcode) {
        //Add kernel.
        case Op_Code::_add: 
            add_kernel<<<blocks, threads>>>(a, b, c);
            break;

        //Subtract kernel.
        case Op_Code::_sub:
            sub_kernel<<<blocks, threads>>>(a, b, c);
            break;
    
        //Element multiplication kernel.
        case Op_Code::_e_mult:
            e_mult_kernel<<<blocks, threads>>>(a, b, c);
            break;

        //Element multiplication kernel.
        case Op_Code::_scale:
            scale_kernel<<<blocks, threads>>>(a, c);
            break;

        //Sigmoid kernel.
        case Op_Code::_sigmoid:
            sigmoid_kernel<<<blocks, threads>>>(a);
            break;

        //Derivitive of sigmoid kernel.
        case Op_Code::_d_sigmoid:
            d_sigmoid_kernel<<<blocks, threads>>>(a);
            break;

        //Relu kernel.
        case Op_Code::_relu:
            relu_kernel<<<blocks, threads>>>(a);
            break;

        //Derivitive of relu kernel.
        case Op_Code::_d_relu:
            d_relu_kernel<<<blocks, threads>>>(a);
            break;

        //Unsupported/Invalid opcode.
        default:
            std::cerr << "Ops_gpu: kernel_call: Error: Unsupported opcode." << std::endl;
            break;
    }
}


}
